#include "hip/hip_runtime.h"
 __global__ void my2DConvKernel(float *d_Result, float *d_Data, int dataW, int dataH)
 {
   // original image based coordinate
   int y = blockIdx.y * blockDim.y   threadIdx.y;
  int x = blockIdx.x * blockDim.x   threadIdx.x;
 
   int  BiasY = y - KERNEL_RADIUS;
   int  BiasX = x - KERNEL_RADIUS;
 
   float sum = 0;
   for(int j = 0; j &lt; KERNEL_LENGTH;   j)
   {
     //out of image range
     if (BiasY   j &lt; 0 || BiasY   j &gt;= dataH) 
       continue;
 
     for(int i = 0; i &lt; KERNEL_LENGTH;   i)
     {
       //out of image range
       if (BiasX   i &lt; 0 || BiasX   i &gt;= dataW) 
         continue;
 
       sum  = d_Data[(BiasY   j) * dataW   BiasX   i] * 
              c_Kernel[KERNEL_LENGTH * j   i];
     }
   }
 
   d_Result[y * dataW   x] = sum;  
 }
