#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void convolution(float* inputImage , float* outputImage, float* filter, int imageWidth, int imageHeight, int filterWidth) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    int i = blockIdx.y * blockDim.y + threadIdx.y; /*raccoon: y = row*/	
    int j = blockIdx.x * blockDim.x + threadIdx.x; /*raccoon: x = column*/
    float sum = 0;
    int halffilterSize = filterWidth / 2;
    int k, l;
    for(k = -halffilterSize; k <= haltfilterSize; k++){
	    for(l = halffilterSize; l <=halffilterSize; l++){
		    if(i + k >= 0 && i + k < imageHeight && j + l >= 0 && j + l < imageWidth){
			    sum += inputImage[(i + k) * imageWidth + j + l] * 
				    filter[(k + halffilterSize) * filterWidth + l + halffilterSize];
		    }
	    }
    }
    outputImage[i * imageWidth + j] = sum;    
}
// Host front-end function that allocates the memory and launches the GPU kernel
//raccoon:img = output
//raccoon:resX = width
//raccoon:resY = height
void hostFE (float* inputImage, float* outputImage, float* filter, int imageWidth, int imageHeight, int filterWidth)
{
    /*------------------raccoon------------------------*/
    size_t imageSize = imageHeight * imageWidth * sizeof(float);
    size_t filterSize = filterWidth * filterWidth * sizeof(float);
    float* source_image;
    float* output_image;
    float* kernel_filter;
    hipMalloc(&source_image, imageSize);
    hipMalloc(&output_image, imageSize);
    hipMalloc(&kernel_filter, filterSize);
    hipMemcpy(source_image, inputImage, imageSize, hipMemcpyHostToDevice);
    hipMemcpy(output_image, outputImage, imageSize, hipMemcpyHostToDevice);
    hipMemcpy(kernel_filter, filter, filterSize, hipMemcpyHostToDevice);
    dim3 dimBlock(8, 8);
    dim3 dimGrid(imageWidth / dimBlock.x, imageHeight / dimBlock.y);
    convolution <<<dimGrid, dimBlock>>>(, lowerY, stepX, stepY, maxIterations, result);
    hipMemcpy(img, result, size, hipMemcpyDeviceToHost);
    hipFree(result);
}
