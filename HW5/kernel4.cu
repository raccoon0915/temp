#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY, int maxIterations, int* result) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    int thisX = blockIdx.x * blockDim.x * 2 + threadIdx.x * 2;
    int thisY = blockIdx.y * blockDim.y * 2 + threadIdx.y * 2;
    //printf("thisX:%d thisY:%d\n", thisX, thisY);
    for(int i = 0; i < 2; i++)
       for(int j = 0; j < 2; j++){
          float x = lowerX + (thisX + i) * stepX;
          float y = lowerY + (thisY + j) * stepY;
          float z_re = x, z_im = y;
          int ii;
          for (ii = 0; ii < maxIterations; ++ii)
          {
             if (z_re * z_re + z_im * z_im > 4.f)
                break;
             float new_re = z_re * z_re - z_im * z_im;
             float new_im = 2.f * z_re * z_im;
             z_re = x + new_re;
             z_im = y + new_im;
          }
          result[(thisY + j) * gridDim.x * blockDim.x * 2 + (thisX + i)] = ii;
    }
    
}

// Host front-end function that allocates the memory and launches the GPU kernel
//raccoon:img = output
//raccoon:resX = width
//raccoon:resY = height
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
    /*------------------raccoon------------------------*/
    size_t size = resX * resY * sizeof(int);
    //int *temp;
    //cudaHostAlloc(&temp, size, cudaHostAllocMapped);
    int *result;
    size_t pitch;
    hipMallocPitch(&result, &pitch, resX * sizeof(int), resY * sizeof(int));
    //cudaMemcpy(result, temp, size, cudaMemcpyHostToDevice);
    dim3 dimBlock(25, 40);
    dim3 dimGrid(resX / dimBlock.x / 2, resY / dimBlock.y / 2);
    //dim3 dimBlock(25,40);
    //dim3 dimGrid(2,2);
    mandelKernel <<<dimGrid, dimBlock>>>(lowerX, lowerY, stepX, stepY, maxIterations, result);
    hipMemcpy(img, result, size, hipMemcpyDeviceToHost);
    hipFree(result);
}
