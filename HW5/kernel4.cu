#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY, int maxIterations, int* result) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    int thisX = (blockIdx.x << 6) + (threadIdx.x << 1);
    int thisY = (blockIdx.y << 4) + (threadIdx.y << 1);
    //printf("thisX:%d thisY:%d\n", thisX, thisY);
    for(int i = 0; i < 2; i++)
       for(int j = 0; j < 2; j++){
          float x = lowerX + (thisX + i) * stepX;
          float y = lowerY + (thisY + j) * stepY;
          float z_re = x, z_im = y;
          int ii;
          for (ii = 0; ii < maxIterations; ++ii)
          {
             if (z_re * z_re + z_im * z_im > 4.f)
                break;
             float new_re = z_re * z_re - z_im * z_im;
             float new_im = 2.f * z_re * z_im;
             z_re = x + new_re;
             z_im = y + new_im;
          }
	  int index = (thisY + j) * gridDim.x * blockDim.x * 2 +(thisX + i);
          result[index] = ii;
    }
    
}

// Host front-end function that allocates the memory and launches the GPU kernel
//raccoon:img = output
//raccoon:resX = width
//raccoon:resY = height
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
    /*------------------raccoon------------------------*/
    size_t size = resX * resY * sizeof(int);
    int *temp;
    hipHostAlloc(&temp, size, hipHostMallocMapped);
    int *result;
    size_t pitch;
    hipMallocPitch(&result, &pitch, resX * sizeof(int), resY * sizeof(int));
    hipMemcpyAsync(result, temp, size, hipMemcpyHostToDevice, 0);
    dim3 dimBlock(32, 8);
    dim3 dimGrid(resX / dimBlock.x / 2, resY / dimBlock.y / 2);
    //dim3 dimBlock(25,40);
    //dim3 dimGrid(2,2);
    mandelKernel <<<dimGrid, dimBlock>>>(lowerX, lowerY, stepX, stepY, maxIterations, result);
    hipMemcpyAsync(img, result, size, hipMemcpyDeviceToHost, 0);
    hipFree(result);
}
