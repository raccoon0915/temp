#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY, int maxIterations, int* result) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;
    float x = lowerX + thisX * stepX;
    float y = lowerY + thisY * stepY;
    float z_re = x, z_im = y;
    int i;
    for (i = 0; i < maxIterations; ++i)
    {

      if (z_re * z_re + z_im * z_im > 4.f)
        break;

      float new_re = z_re * z_re - z_im * z_im;
      float new_im = 2.f * z_re * z_im;
      z_re = x + new_re;
      z_im = y + new_im;
    }
    result[thisY * gridDim.x * blockDim.x + thisX] = i;
}

// Host front-end function that allocates the memory and launches the GPU kernel
//raccoon:img = output
//raccoon:resX = width
//raccoon:resY = height
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
    /*------------------raccoon------------------------*/
    size_t size = resX * resY * sizeof(int);
    int *temp = (int*)malloc(size);
    int* result;
    hipMalloc(&result, size);
    dim3 dimBlock(40, 25);
    dim3 dimGrid(resX / dimBlock.x, resY / dimBlock.y);
    mandelKernel <<<dimGrid, dimBlock>>>(lowerX, lowerY, stepX, stepY, maxIterations, result);
    hipMemcpyAsync(img, result, size, hipMemcpyDeviceToHost, 0);
    hipFree(result);
}
