#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY, int maxIterations, float* result) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    int thisX = blockIdx.x * 40 + threadIdx.x;
    int thisY = blockIdx.y * 30 + threadIdx.y;
    float x = lowerX + thisX * stepX;
    float y = lowerY + thisY * stepY;
    float z_re = x, z_im = y;
    int i;
    for (i = 0; i < maxIterations; ++i)
    {

      if (z_re * z_re + z_im * z_im > 4.f)
        break;

      float new_re = z_re * z_re - z_im * z_im;
      float new_im = 2.f * z_re * z_im;
      z_re = x + new_re;
      z_im = y + new_im;
    }
    result[thisY * 1600 + thisX] = i;
}

// Host front-end function that allocates the memory and launches the GPU kernel
//raccoon:img = output
//raccoon:resX = width
//raccoon:resY = height
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
    /*------------------raccoon------------------------*/
    size_t size = resX * resY * sizeof(float);
    float *result = hipMalloc(&result, size);
    dim3 dimBlock(40, 30);
    dim3 dimGrid(resX / dimBlock.x, resY / dimBlock.y);
    mandleKernel <<<dimGrid, dimBlock>>>(lowerX, lowerY, stepX, stepY, maxIterations, result);
    hipMemcpy(img, result, size, hipMemcpyDeviceToHost);
}
